
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Hello World from thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main()
{
    // Define grid and block dimensions
    int blocks = 10;
    int threadsPerBlock = 5;

    // Launch the kernel
    helloCUDA<<<blocks, threadsPerBlock>>>();
    // cuda_hello<<<blocks, blocks>>>();

    // Wait for GPU to finish before exiting
    hipDeviceSynchronize();

    // Check for errors
    // cudaError_t error = cudaGetLastError();
    // if (error != cudaSuccess)
    // {
    //     fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(error));
    //     return 1;
    // }

    return 0;
}
